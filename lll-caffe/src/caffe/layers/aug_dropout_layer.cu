#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/aug_dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void AugDropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
__global__ void AugDropoutForward_Copymask(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = static_cast<Dtype>(mask[index]);
  }
}

template <typename Dtype>
__global__ void AugDropoutForwardV2(const int n, const Dtype* in,
    const Dtype* assigned_mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (static_cast<unsigned int>(assigned_mask[index]) > threshold) * scale;
  }
}

template <typename Dtype>
void AugDropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);

    const Dtype* assigned_mask=NULL;
    Dtype* top_mask_data=NULL;
    if(assigned_mask_){
        assigned_mask = bottom[1]->gpu_data();
    }
    if(top.size() > 1){
        top_mask_data = top[1]->mutable_gpu_data();
    }

    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    if(assigned_mask_){
        AugDropoutForwardV2<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, bottom_data, assigned_mask, uint_thres_, scale_, top_data);
    }
    else{
        AugDropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, bottom_data, mask, uint_thres_, scale_, top_data);
        if(top.size() > 1){
            AugDropoutForward_Copymask<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
                count, bottom_data, mask, uint_thres_, scale_, top_mask_data);
        }
    }
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
  }
}

template <typename Dtype>
__global__ void AugDropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
__global__ void AugDropoutBackwardV2(const int n, const Dtype* in_diff,
    const Dtype* assigned_mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (static_cast<unsigned int>(assigned_mask[index]) > threshold);
  }
}

template <typename Dtype>
void AugDropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      const Dtype* assigned_mask=NULL;
      if(assigned_mask_){
          assigned_mask = bottom[1]->gpu_data();
      }
      // NOLINT_NEXT_LINE(whitespace/operators)
      if(assigned_mask_){
          AugDropoutBackwardV2<Dtype><<<CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS>>>(
              count, top_diff, assigned_mask, uint_thres_, scale_, bottom_diff);
      }
      else{
          AugDropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS>>>(
              count, top_diff, mask, uint_thres_, scale_, bottom_diff);
      }
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AugDropoutLayer);

}  // namespace caffe
