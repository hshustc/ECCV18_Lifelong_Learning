#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/aug_sigmoid_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void AugSigmoidForward(const int n, const Dtype* in, Dtype* out, Dtype multiplier) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 0.5 * tanh(0.5 * in[index]) + 0.5;
    out[index] = out[index] * multiplier;
  }
}

template <typename Dtype>
void AugSigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype multiplier = this->layer_param_.aug_sigmoid_param().multiplier();
  // NOLINT_NEXT_LINE(whitespace/operators)
  AugSigmoidForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, multiplier);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void AugSigmoidBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, Dtype* out_diff, Dtype multiplier) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index] / multiplier;
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x) * multiplier;
  }
}

template <typename Dtype>
void AugSigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype multiplier = this->layer_param_.aug_sigmoid_param().multiplier();
    // NOLINT_NEXT_LINE(whitespace/operators)
    AugSigmoidBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_diff, multiplier);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(AugSigmoidLayer);


}  // namespace caffe
